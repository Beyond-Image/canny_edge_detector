#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vector_add_kernel(int* A, int* B, int* C, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < N) {
        C[index] = A[index] + B[index];
    }
}

int main() {
    const int N = 10;
    int A[N], B[N], C[N];

    // Initialize input arrays
    for (int i = 0; i < N; i++) {
        A[i] = i;
        B[i] = i * 2;
    }

    int* d_A, * d_B, * d_C;

    // Allocate device memory
    hipMalloc((void**)&d_A, N * sizeof(int));
    hipMalloc((void**)&d_B, N * sizeof(int));
    hipMalloc((void**)&d_C, N * sizeof(int));

    // Copy input data from host to device
    hipMemcpy(d_A, A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with one block and N threads
    vector_add_kernel <<<1, N >>> (d_A, d_B, d_C, N);

    // Copy result from device to host
    hipMemcpy(C, d_C, N * sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    std::cout << "Resulting vector C:\n";
    for (int i = 0; i < N; i++) {
        std::cout << C[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}